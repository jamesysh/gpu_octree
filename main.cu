#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

//typedef unsigned _int64 uint64_t




int main(){

 
    uint64_t key[5] = {234,5938,23,94,55};
    int index[5] = {0,1,2,3,4}; 
    
    uint64_t* key_d;
    int* index_d;
    hipMalloc((void**)&key_d,sizeof(uint64_t)*5);
    hipMalloc((void**)&index_d,sizeof(int)*5);
    hipMemcpy(key_d,key,sizeof(uint64_t)*5,hipMemcpyHostToDevice);
    hipMemcpy(index_d,index,sizeof(int)*5,hipMemcpyHostToDevice);

    thrust::sort_by_key(thrust::device,key_d,key_d+5,index_d);   

    
    hipFree(key_d);
    hipFree(index_d);



}
